
#include <hip/hip_runtime.h>
#include <getopt.h>
#include <stddef.h>
#include <stdio.h>
#include <memory.h>
#include <stdlib.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>(); 
    return 0;
}